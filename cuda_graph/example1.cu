#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <sstream>

// A simple kernel that adds two vectors
__global__ void vectorAdd(const int *A, const int *B, int *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

// Function to export CUDA graph to a DOT file
void exportGraphToDot(hipGraph_t graph, const char* filename) {
    std::ofstream file(filename);
    if (file.is_open()) {
        file << "digraph CUDA_Graph {\n";

        size_t numNodes;
        hipGraphNode_t* nodes;

        // Get nodes from the graph
        hipGraphGetNodes(graph, NULL, &numNodes);
        nodes = new hipGraphNode_t[numNodes];
        hipGraphGetNodes(graph, nodes, &numNodes);

        for (size_t i = 0; i < numNodes; ++i) {
            hipGraphNode_t node = nodes[i];

            hipKernelNodeParams kernelNodeParams;
            hipMemcpyNodeParams memcpyNodeParams;

            hipGraphNodeType nodeType;
            hipGraphNodeGetType(node, &nodeType);

            std::stringstream ss;
            ss << "Node" << i;

            // Identify the node type
            if (nodeType == hipGraphNodeTypeKernel) {
                hipGraphKernelNodeGetParams(node, &kernelNodeParams);
                file << "  " << ss.str() << " [label=\"Kernel Node\"];\n";
            } else if (nodeType == hipGraphNodeTypeMemcpy) {
                hipGraphMemcpyNodeGetParams(node, &memcpyNodeParams);
                file << "  " << ss.str() << " [label=\"Memcpy Node\"];\n";
            }

            // Get dependencies and add edges
            size_t numDependencies;
            hipGraphNode_t* dependencies;
            hipGraphNodeGetDependencies(node, NULL, &numDependencies);
            dependencies = new hipGraphNode_t[numDependencies];
            hipGraphNodeGetDependencies(node, dependencies, &numDependencies);

            for (size_t j = 0; j < numDependencies; ++j) {
                std::stringstream ssDep;
                ssDep << "Node" << i << " -> Node" << j << ";\n";
                file << "  " << ssDep.str();
            }

            delete[] dependencies;
        }

        delete[] nodes;

        file << "}\n";
        file.close();
    }
}

int main() {
    // Vector size
    int N = 1 << 20;
    size_t size = N * sizeof(int);

    // Allocate input vectors A and B in host memory
    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate vectors in device memory
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Create a CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Create a CUDA graph
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    // Begin recording graph
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // Copy vectors from host memory to device memory
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);

    // End recording graph
    hipStreamEndCapture(stream, &graph);

    // Export the graph to a DOT file
    exportGraphToDot(graph, "cuda_graph.dot");

    // Instantiate and launch the graph
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    hipGraphLaunch(graphExec, stream);

    // Wait for the graph execution to finish
    hipStreamSynchronize(stream);

    // Check the result for correctness
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            printf("Mismatch at index %d: %d != %d + %d\n", i, h_C[i], h_A[i], h_B[i]);
            break;
        }
    }

    // Clean up
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipStreamDestroy(stream);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Completed successfully.\n");
    return 0;
}
