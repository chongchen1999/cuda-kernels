#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }

    if (blockDim.x < 256) {
        __shared__ float shared_A[256];
        shared_A[threadIdx.x] = A[idx];
    } else {
        __shared__ float shared_A[512];
        shared_A[threadIdx.x] = A[idx];
    }
}

int main() {
    int n = 1000000; // Size of vectors
    size_t size = n * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < n; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Number of threads in each thread block
    int threadsPerBlock = 256;

    // Number of thread blocks in grid
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector addition kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < n; i++) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << "!\n";
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED\n";

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
