#include "../utils/includes/cpu_random.h"
#include "includes/matrix_transpose_v1.cuh"
#include "includes/matrix_transpose_v2.cuh"
#include "includes/matrix_transpose_v3.cuh"
#include "includes/matrix_transpose_v4.cuh"
#include <iostream>

static const int M = 4096;
static const int seq_len = 5000;
static const int iters = 1000;

void printMatrix(float *A, int M, int N) {
    puts("");
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%.2f ", A[i * N + j]);
        }
        puts("");
    }
    puts("");
}

bool checkResult(float *A, float *B, int size) {
    for (int i = 0; i < size; ++i) {
        if (A[i] - B[i] > 1e-6) {
            return false;
        }
    }
    return true;
}

int main() {
    auto host_A = std::make_unique<float[]>(seq_len * M);
    auto host_AT_cpu = std::make_unique<float[]>(seq_len * M);
    auto host_AT1 = std::make_unique<float[]>(seq_len * M);
    auto host_AT2 = std::make_unique<float[]>(seq_len * M);
    auto host_AT3 = std::make_unique<float[]>(seq_len * M);
    auto host_AT4 = std::make_unique<float[]>(seq_len * M);
    randomTools::fastRandomFill<float>(host_A.get(), seq_len * M, 0.f, 1.f);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < seq_len; j++) {
            host_AT_cpu[j * M + i] = host_A[i * seq_len + j];
        }
    }

    float *device_A, *device_AT;
    hipMalloc(reinterpret_cast<void **>(&device_A), seq_len * M * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&device_AT), seq_len * M * sizeof(float));
    hipMemcpy(device_A, host_A.get(), seq_len * M * sizeof(float), hipMemcpyHostToDevice);

    matrixTransposeV1::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    hipMemcpy(host_AT1.get(), device_AT, seq_len * M * sizeof(float), hipMemcpyDeviceToHost);

    matrixTransposeV2::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    hipMemcpy(host_AT2.get(), device_AT, seq_len * M * sizeof(float), hipMemcpyDeviceToHost);

    matrixTransposeV3::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    hipMemcpy(host_AT3.get(), device_AT, seq_len * M * sizeof(float), hipMemcpyDeviceToHost);

    matrixTransposeV4::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    hipMemcpy(host_AT4.get(), device_AT, seq_len * M * sizeof(float), hipMemcpyDeviceToHost);

    puts("");
    matrixTransposeV1::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    matrixTransposeV2::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    matrixTransposeV3::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);
    matrixTransposeV4::launchTranspose<float>(device_A, device_AT, M, seq_len, iters);

    std::cout << checkResult(host_AT_cpu.get(), host_AT1.get(), seq_len * M) << std::endl;
    std::cout << checkResult(host_AT_cpu.get(), host_AT2.get(), seq_len * M) << std::endl;
    std::cout << checkResult(host_AT_cpu.get(), host_AT3.get(), seq_len * M) << std::endl;
    std::cout << checkResult(host_AT_cpu.get(), host_AT4.get(), seq_len * M) << std::endl;

    hipFree(device_A);
    hipFree(device_AT);
    return 0;
}