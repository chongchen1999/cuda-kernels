#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

const int N = 1 << 25;
const int iterations = 2000;

__global__ void add_v2(float *a, float *b, float *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    result[tid] = a[tid] + b[tid];
}

bool checkResult(float *a, float *b) {
    for (int i = 0; i < N; ++i) {
        if (a[i] != b[i]) {
            return false;
        }
    }
    return true;
}

int main() {
    float *host_a = (float *) malloc(N * sizeof(float));
    float *host_b = (float *) malloc(N * sizeof(float));
    float *host_result = (float *) malloc(N * sizeof(float));
    float *cpu_result = (float *) malloc(N * sizeof(float));

    float *device_a;
    float *device_b;
    float *device_result;

    hipMalloc((void **) &device_a, N * sizeof(float));
    hipMalloc((void **) &device_b, N * sizeof(float));
    hipMalloc((void **) &device_result, N * sizeof(float));

    for (int i = 0; i < N; ++i) {
        host_a[i] = i;
        host_b[i] = (N % (i + 1)) * 1.0f;
        cpu_result[i] = host_a[i] + host_b[i];
    }

    hipMemcpy(device_a, host_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, N * sizeof(float), hipMemcpyHostToDevice);
    int block_size = 256;
    int grid_size = (N - 1) / block_size + 1;

    dim3 Grid(grid_size);
    dim3 Block(block_size);

    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iterations; ++i) {
        add_v2<<<Grid, Block>>>(device_a, device_b, device_result);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(host_result, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    if (!checkResult(host_result, cpu_result)) {
        printf("Wrong Answer!\n");
    } else {
        printf("Success\n");
    }

    // Calculate memory bandwidth
    double total_data_transfer = 3 * N * sizeof(float); // 2 input arrays and 1 output array
    double bandwidth = (total_data_transfer / (1 << 30)) / (milliseconds / 1000.0 / iterations); // GB/s
    std::cout << "Elapsed time: " << milliseconds / 1000.0 << " seconds" << std::endl;
    std::cout << "Memory Bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
    free(host_a);
    free(host_b);
    free(host_result);
    free(cpu_result);

    return 0;
}