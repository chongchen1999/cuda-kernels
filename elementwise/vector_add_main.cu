#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../utils/includes/cpu_random.h"
#include "includes/vector_add_v5.cuh"
#include "includes/vector_add_v6.cuh"

static const int iters = 1000;

void cublas_vector_add(int n, float *d_A, float *d_B, float alpha) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iters; ++i) {
        hipblasSaxpy(handle, n, &alpha, d_A, 1, d_B, 1);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cublas time: %.4f ms!\n", milliseconds / iters);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    float alpha = 1.0f;
    int n = 1 << 25;
    size_t size = n * sizeof(float);
    // Host vectors
    float *h_A = static_cast<float *>(malloc(size));
    float *h_B = static_cast<float *>(malloc(size));

    // Initialize vectors
    randomTools::fastRandomFill(h_A, n, 0.f, 1.f);
    randomTools::fastRandomFill(h_B, n, 0.f, 1.f);

    // Device vectors
    float *d_A_cublas, *d_B_cublas;
    hipMalloc(reinterpret_cast<void **>(&d_A_cublas), size);
    hipMalloc(reinterpret_cast<void **>(&d_B_cublas), size);
    hipMemcpy(d_A_cublas, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B_cublas, h_B, size, hipMemcpyHostToDevice);

    float *d_A_mykernel, *d_B_mykernel;
    hipMalloc(reinterpret_cast<void **>(&d_A_mykernel), size);
    hipMalloc(reinterpret_cast<void **>(&d_B_mykernel), size);
    hipMemcpy(d_A_mykernel, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B_mykernel, h_B, size, hipMemcpyHostToDevice);

    // Perform vector addition: B = alpha * A + B
    cublas_vector_add(n, d_A_cublas, d_B_cublas, alpha);
    my_vector_add_vecsize1::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    my_vector_add_vecsize4::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    cublas_vector_add(n, d_A_cublas, d_B_cublas, alpha);
    my_vector_add_vecsize1::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    my_vector_add_vecsize4::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    cublas_vector_add(n, d_A_cublas, d_B_cublas, alpha);
    my_vector_add_vecsize1::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    my_vector_add_vecsize4::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    cublas_vector_add(n, d_A_cublas, d_B_cublas, alpha);
    my_vector_add_vecsize1::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);
    my_vector_add_vecsize4::launchVectorAdd(n, d_A_mykernel, d_B_mykernel, alpha, iters);

    // Copy result back to host
    // hipMemcpy(h_B, d_B_cublas, size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A_cublas);
    hipFree(d_B_cublas);
    free(h_A);
    free(h_B);
    hipFree(d_A_mykernel);
    hipFree(d_B_mykernel);

    return 0;
}
