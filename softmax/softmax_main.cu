#include "includes/softmax_v1.cuh"
#include "includes/softmax_v2.cuh"
#include "includes/softmax_v3.cuh"
#include <iostream>
#include <memory>
#include <algorithm>
#include "../utils/includes/cpu_random.h"
#include "includes/softmax_cudnn.h"
#include "includes/softmax_cpu.h"
#include "../utils/includes/check_result.h"

int main() {
    const int M = 1000;
    const int N = 2048;
    float *host_data, *device_data;
    host_data = static_cast<float *>(malloc(sizeof(float) * N * M));
    hipMalloc(reinterpret_cast<void **>(&device_data), sizeof(float) * N * M);

    randomTools::randomFill<float>(host_data, M * N, .0f, 1.0f);
    hipMemcpy(device_data, host_data, M * N * sizeof(float), hipMemcpyHostToDevice);

    float *host_output_block_based, *host_output_warp_based, *device_output_block_based, *device_output_warp_based;
    host_output_block_based = static_cast<float *>(malloc(M * N * sizeof(float)));
    host_output_warp_based = static_cast<float *>(malloc(M * N * sizeof(float)));
    hipMalloc(reinterpret_cast<void **>(&device_output_block_based), M * N * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&device_output_warp_based), M * N * sizeof(float));

    float *host_cudnn_output, *cudnn_output;
    host_cudnn_output = static_cast<float *>(malloc(M * N * sizeof(float)));
    hipMalloc(reinterpret_cast<void **>(&cudnn_output), M * N * sizeof(float));

    int times = 1000;

    std::cout << "Start cuDNN!" << std::endl;
    cuDNN::launchSoftmax(device_data, cudnn_output, M, N, times);
    hipMemcpy(host_cudnn_output, cudnn_output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "cuDNN Done!" << std::endl << std::endl;

    std::cout << "Start block-register based softmax!" << std::endl;
    blockBasedSoftmax_register::launchSoftmax(device_data, device_output_block_based, M, N, times);
    hipMemcpy(host_output_block_based, device_output_block_based, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Block-register based softmax Done!" << std::endl << std::endl;

    std::cout << "Start block-shared memory based softmax!" << std::endl;
    blockBasedSoftmax_shared::launchSoftmax(device_data, device_output_block_based, M, N, times);
    hipMemcpy(host_output_block_based, device_output_block_based, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Block-shared memory based softmax Done!" << std::endl << std::endl;

    std::cout << "Start warp based softmax!" << std::endl;
    warpBasedSoftmax::launchSoftmax(device_data, device_output_warp_based, M, N, times);
    hipMemcpy(host_output_warp_based, device_output_warp_based, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Warp based softmax Done!" << std::endl << std::endl;

    float *cpu_result = static_cast<float *>(malloc(M * N * sizeof(float)));
    cpu_softmax::launchSoftmax(cpu_result, host_data, M, N);

    if (checkResult(host_output_block_based, host_cudnn_output, N)) {
        std::cout << "Block Test passed!" << std::endl;
    } else {
        std::cout << "Block Test failed!" << std::endl;
    }

    if (checkResult(host_output_warp_based, host_cudnn_output, N)) {
        std::cout << "Warp Test passed!" << std::endl;
    } else {
        std::cout << "Warp Test failed!" << std::endl;
    }

    hipFree(device_data);
    hipFree(device_output_block_based);
    hipFree(cudnn_output);
    hipFree(host_output_block_based);
    hipFree(device_output_warp_based);
    free(host_data);
    free(host_output_block_based);
    free(host_cudnn_output);
    free(cpu_result);
    free(host_output_warp_based);

    return 0;
}