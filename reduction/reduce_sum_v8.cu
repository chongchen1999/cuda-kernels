#include <hip/hip_runtime.h>
#include <iostream>
#include <hipcub/hipcub.hpp>  // CUB library header

int main() {
    int size = 1 << 25; // Size of the array
    int bytes = size * sizeof(float);
    int iterations = 1000;

    // Host memory
    float *host_data = new float[size];
    float host_result;

    // Initialize input data
    for (int i = 0; i < size; i++) {
        host_data[i] = 1.0f; // Example data
    }

    // Device memory
    float *device_data, *device_result;
    hipMalloc(&device_data, bytes);
    hipMalloc(&device_result, sizeof(float));

    // Copy data to device
    hipMemcpy(device_data, host_data, bytes, hipMemcpyHostToDevice);

    // CUB reduction
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, device_data, device_result, size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iterations; i++) {
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, device_data, device_result, size);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds / iterations);

    // Copy result back to host
    hipMemcpy(&host_result, device_result, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Sum: " << host_result << std::endl;

    // Free memory
    hipFree(device_data);
    hipFree(device_result);
    hipFree(d_temp_storage);
    delete[] host_data;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}