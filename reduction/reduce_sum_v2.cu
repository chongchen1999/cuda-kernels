#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

const int seq_len = 1 << 25; // 2^25 elements
const int iterations = 5000;

template <int block_size>
__global__ void sum_kernel(int *data, int *partial_sums) {
    __shared__ int shared_data[block_size];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int offset = blockDim.x * gridDim.x;

    int sum = 0;
    for (int i = tid; i < seq_len; i += offset) {
        sum += data[i];
    }
    shared_data[threadIdx.x] = sum;
    __syncthreads();

    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        partial_sums[blockIdx.x] = shared_data[0];
    }
}

void get_sum(const int *data, const int &N, int &sum) {
    for (int i = 0; i < N; ++i) {
        sum += data[i];
    }
}

int main() {
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    int *host_data = (int *)malloc(seq_len * sizeof(int));
    int cpu_sum = 0;
    for (int i = 0; i < seq_len; ++i) {
        int random_int = std::rand() % 57;
        host_data[i] = random_int;
        cpu_sum += random_int;
    }
    printf("CPU sum: %d\n", cpu_sum);

    int *device_data;
    hipMalloc(&device_data, seq_len * sizeof(int));

    constexpr int block_size = 256;
    constexpr int grid_size = (seq_len + block_size - 1) / block_size;

    dim3 block(block_size);
    dim3 grid(grid_size);

    int *host_partial_sums = (int *)malloc(grid_size * sizeof(int));
    int *device_partial_sums;
    hipMalloc(&device_partial_sums, grid_size * sizeof(int));

    hipMemcpy(device_data, host_data, seq_len * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iterations; ++i) {
        sum_kernel<block_size><<<grid, block>>>(device_data, device_partial_sums);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds / iterations);

    hipMemcpy(host_partial_sums, device_partial_sums, grid_size * sizeof(int), hipMemcpyDeviceToHost);
    int gpu_sum = 0;
    get_sum(host_partial_sums, grid_size, gpu_sum);
    if (cpu_sum != gpu_sum) {
        printf("Error: %d != %d\n", cpu_sum, gpu_sum);
    } else {
        printf("Success!\n");
    }

    // Calculate Bandwidth
    float total_data_transferred = seq_len * sizeof(int) + grid_size * sizeof(int); // in bytes
    float average_time_per_iteration = milliseconds / iterations / 1000; // in seconds
    float bandwidth = total_data_transferred / average_time_per_iteration / (1 << 30); // in GB/s

    printf("Bandwidth: %f GB/s\n", bandwidth);

    hipFree(device_data);
    hipFree(device_partial_sums);
    free(host_data);
    free(host_partial_sums);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
