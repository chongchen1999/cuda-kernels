// wmma + pipeline

#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>

const int bm = 128;
const int bn = 128;
const int bk = 32;

const int wm = 64;
const int wn = 64;
const int wk = 16;

const int wmma_m = 16;
const int wmma_n = 16;
const int wmma_k = 16;

__device__ __forceinline__ void loadSmemA(half *smem, half *A, int M, int K, int k) {
    // load 128 * 32
    const int by = blockIdx.y;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int row = (i << 5) + (tid >> 2); // 1 thread load 128-bit, 4 threads per row
        const int col = (lane_id >> 2) << 3; // 128-bit per thread, aka 8 half per thread

        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        void *ptr = reinterpret_cast<void *>(smem + (row_o << 9) + (col_o << 8) + (row_i << 4) + col_i);
        uint32_t smem_ptr;

        asm(
            "{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, smem_ptr; }\n"
            : "=r"(smem_ptr)
            : "l"(ptr)
        );

        asm volatile(
            "cp.async.cg.shared.global [%0], [%1], %2;\n"
            :
            : "r"(smem_ptr), "l"(&A[(by * bm + row) * K + (k * bk + col)]), "n"(16)
        );
    }
}

__device__ __forceinline__ void loadSmemB(half *smem, half *B, int N, int K, int k) {
    // load 128 * 32
    const int bx = blockIdx.x;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    // const int tid = warp_y * 64 + warp_x * 32 + lane_id;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int row = (i << 5) + (tid >> 2); // 1 thread load 128-bit, 4 threads per row
        const int col = (lane_id >> 2) << 3; // 128-bit per thread, aka 8 half per thread

        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        void *ptr = reinterpret_cast<void *>(smem + (row_o << 9) + (col_o << 8) + (row_i << 4) + col_i);
        uint32_t smem_ptr;

        asm(
            "{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, smem_ptr; }\n"
            : "=r"(smem_ptr)
            : "l"(ptr)
        );

        asm volatile(
            "cp.async.cg.shared.global [%0], [%1], %2;\n" 
            :
            : "r"(smem_ptr), "l"(&B[(bx * bn + row) * K + (k * bk + col)]), "n"(16)
        );
    }
}

__device__ __forceinline__ void loadSmemC(float *smem, half *C, int M, int N) {
    // load 128 * 128
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < bm; ++i) {
        const int row = i;
        const int col = tid;

        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        smem[(row_o << 9) + (col_o << 8) + (row_i << 4) + col_i] = 
            static_cast<float>(C[(by * bm + row) * N + bx * bn + col]);
    }
}

__device__ __forceinline__ void storeSmemC(half *C, float *smem, int M, int N) {
    // load 128 * 128
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < bm; ++i) {
        const int row = i;
        const int col = tid;

        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        C[(by * bm + row) * N + bx * bm + col] = 
            static_cast<half>(smem[(row_o << 9) + (col_o << 8) + (row_i << 4) + col_i]);
    }
}

__device__ __forceinline__ void loadFragA(
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmma_m, wmma_n, wmma_k, half, nvcuda::wmma::row_major> *frag, 
    half *smem, 
    int k
) {
    // load 64x16
    const int warp_y = threadIdx.z;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int row = (warp_y << 6) + (i << 4);
        const int col = k * wk;
        nvcuda::wmma::load_matrix_sync(
            frag[i], 
            smem + ((row >> 4) << 9) + ((col >> 4) << 8), 
            16
        );
    }
}

__device__ __forceinline__ void loadFragB(
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmma_m, wmma_n, wmma_k, half, nvcuda::wmma::col_major> *frag, 
    half *smem, 
    int ki
) {
    // load 64x16
    int warp_x = threadIdx.y;
    for (int i = 0; i < 4; ++i) {
        const int row = (warp_x << 6) + (i << 4);
        const int col = ki * wk;
        nvcuda::wmma::load_matrix_sync(
            frag[i], 
            smem + ((row >> 4) << 9) + ((col >> 4) << 8), 
            16
        );
    }
}

__device__ __forceinline__ void storeAccum(
    float *ptr, 
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmma_m, wmma_n, wmma_k, float> *frag
) {
    // store 64x64
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            const int row = (warp_y << 6) + (i << 4);
            const int col = (warp_x << 6) + (j << 4);

            // laoyut: [8, 8, 16, 16]
            nvcuda::wmma::store_matrix_sync(
                ptr + ((row >> 4) << 9) + ((col >> 4) << 8), 
                frag[(i << 2) + j], 16, 
                nvcuda::wmma::mem_row_major
            );
        }
    }
}

__device__ __forceinline__ void warpMma(
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmma_m, wmma_n, wmma_k, half, nvcuda::wmma::row_major> *frag_a, 
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmma_m, wmma_n, wmma_k, half, nvcuda::wmma::col_major> *frag_b, 
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmma_m, wmma_n, wmma_k, float> *accum,
    half *SA, 
    half *SB,
    const int inner_iters_k,
    const int frags_m,
    const int frags_n
) {
    #pragma unroll
    for (int k = 0; k < inner_iters_k; ++k) {
        // 64x64x16 mma for each warp
        loadFragA(frag_a, SA, k);
        loadFragB(frag_b, SB, k);

        #pragma unroll
        for (int i = 0; i < frags_m; ++i) {
            #pragma unroll
            for (int j = 0; j < frags_n; ++j) {
                // 16x16x16 for each wmma
                nvcuda::wmma::mma_sync(
                    accum[i * frags_n + j], 
                    frag_a[i], frag_b[j], 
                    accum[i * frags_n + j]
                );
            }
        }
    }
}

__device__ __forceinline__ void loadSmemAndCommit(
    half *SA, 
    half *SB, 
    half *A, 
    half *B, 
    const int k,
    const int M, 
    const int N, 
    const int K
) {
    loadSmemA(SA, A, M, K, k);
    loadSmemB(SB, B, N, K, k);
    asm volatile("cp.async.commit_group;\n" ::);
}

__global__ void matmul(
    half *A, half *B, half *C, 
    int M, int N, int K, 
    float alpha, float beta
) {
    // A is row-major
    // B is col-major
    // 128 threads [x, y, z] = [32, 2, 2]
    // threadblock mma: 128x128x32
    // warp mma: 64x64x16
    extern __shared__ uint8_t shared_storage[];
    half *SA1 = reinterpret_cast<half *>(shared_storage);
    half *SA2 = SA1 + bm * bk;
    half *SA3 = SA2 + bm * bk;

    half *SB1 = SA3 + bm * bk;
    half *SB2 = SB1 + bn * bk;
    half *SB3 = SB2 + bn * bk;
    float *SC = reinterpret_cast<float *>(shared_storage);

    const int frags_m = wm / wmma_m;
    const int frags_n = wn / wmma_n;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmma_m, wmma_n, wmma_k, half, nvcuda::wmma::row_major> frag_a[frags_m];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmma_m, wmma_n, wmma_k, half, nvcuda::wmma::col_major> frag_b[frags_n];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmma_m, wmma_n, wmma_k, float> accum[frags_m * frags_n];

    for (int i = 0; i < frags_m * frags_n; ++i) {
        nvcuda::wmma::fill_fragment(accum[i], 0.0);
    }
    
    // prologue
    loadSmemAndCommit(SA1, SB1, A, B, 0, M, N, K);
    loadSmemAndCommit(SA2, SB2, A, B, 1, M, N, K);

    const int outter_iters_k = K / bk;
    const int inner_iters_k = bk / wk;

    #pragma unroll
    for (int ko = 0; ko + 3 < outter_iters_k; ko += 3) {
        asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
        __syncthreads();
        if (ko + 2 < outter_iters_k) {
            loadSmemAndCommit(SA3, SB3, A, B, ko + 2, M, N, K);
        }
        warpMma(frag_a, frag_b, accum, SA1, SB1, inner_iters_k, frags_m, frags_n);

        asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
        __syncthreads();
        if (ko + 3 < outter_iters_k) {
            loadSmemAndCommit(SA1, SB1, A, B, ko + 3, M, N, K);
        }
        warpMma(frag_a, frag_b, accum, SA2, SB2, inner_iters_k, frags_m, frags_n);

        asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
        __syncthreads();
        if (ko + 4 < outter_iters_k) {
            loadSmemAndCommit(SA2, SB2, A, B, ko + 4, M, N, K);
        }
        warpMma(frag_a, frag_b, accum, SA3, SB3, inner_iters_k, frags_m, frags_n);
    }

    // the last 3 iterations
    {
        int ko = (outter_iters_k / 3 - 1) * 3;

        if (ko < outter_iters_k) {
            warpMma(frag_a, frag_b, accum, SA1, SB1, inner_iters_k, frags_m, frags_n);
        }
        if (ko + 1 < outter_iters_k) {
            warpMma(frag_a, frag_b, accum, SA2, SB2, inner_iters_k, frags_m, frags_n);
        }
    }

    storeAccum(SC, accum);
    __syncthreads();
    storeSmemC(C, SC, M, N);
}