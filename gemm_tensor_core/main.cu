#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <chrono>
#include <string>
#include <cassert>
#include "includes/commons.cuh"

int STAGES = 1;
int MULTI_THREADING = 1;

extern __global__ void matmul(
    half *A, half *B, half *C, 
    int M, int N, int K, 
    float alpha, float beta
);

float alpha = 1.0;
float beta = 0.0;

void cpuMatmul(half *C, half *A, half *B, int M, int N, int K) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += __half2float(A[i * K + k]) * __half2float(B[j * M + k]);
            }
            C[i * N + j] = __float2half(sum);
        }
    }
}

bool checkResult(half *cpu_result, half *gpu_result, int M, int N, const float eps = 1e-3) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            const float abs_diff = fabs(__half2float(cpu_result[i * N + j]) - __half2float(gpu_result[i * N + j]));
            const float rel_diff = abs_diff / __half2float(cpu_result[i * N + j]);
            if (abs_diff > eps && rel_diff > eps) {
                std::cout << "Mismatch at (" << i << ", " << j << ") CPU = " 
                    << __half2float(cpu_result[i * N + j]) << ", GPU = " 
                    << __half2float(gpu_result[i * N + j]) << "\n";
                return false;
            }
        }
    }
    return true;
}

extern void cublasMatmul(half *C, half *A, half *B, int M, int N, int K);

int main(int argc, char *argv[]) {
    if (argc > 1) {
        assert((argc - 1) % 2 == 0);
        for (int i = 1; i < argc; i += 2) {
            char *key = argv[i];
            char *value = argv[i + 1];
            std::string keys(key);
            if (keys == "stages")
            {
                STAGES = std::atoi(value);
                std::cout << "Setting to " << STAGES << " stages.\n";
            }
            else if (keys == "multi_threading")
            {
                MULTI_THREADING = std::atoi(value);
                std::cout << "Setting to " << MULTI_THREADING << "x threading.\n";
            }
            else if (keys == "iters") {
                iterations = std::atoi(value);
                std::cout << "Testing iters = " << iterations << ".\n";
            }
        }
    }

    srand(time(NULL));
    half *hA = (half *)malloc(M * K * 2);
    half *hB = (half *)malloc(K * N * 2);
    half *hC = (half *)malloc(M * N * 2);
    half *h_cublas = (half *)malloc(M * N * 2);

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            hA[i * K + j] = __float2half(randFP32());
        }
        for (int j = 0; j < N; ++j) {
            hC[i * N + j] = (float)(0);
            h_cublas[i * N + j] = (float)(0);
        }
    }

    for (int k = 0; k < K; ++k) {
        for (int n = 0; n < N; ++n) {
            hB[n * K + k] = __float2half(randFP32());
        }
    }

    // cpuMatmul(cpu_result, hA, hB, M, N, K);
    cublasMatmul(h_cublas, hA, hB, M, N, K);

    puts("CPU matmul done!\n");

    half *dA, *dB, *dC;

    CUDA_CHECK(hipMalloc(&dA, M * K * 2));
    CUDA_CHECK(hipMalloc(&dB, K * N * 2));
    CUDA_CHECK(hipMalloc(&dC, M * N * 2));

    CUDA_CHECK(hipMemcpy(dA, hA, M * K * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, K * N * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dC, hC, M * N * 2, hipMemcpyHostToDevice));

    dim3 dimBlock(block_tile_k, 2 * MULTI_THREADING, 2);
    dim3 dimGrid((N + block_tile_n - 1) / block_tile_n, (M + block_tile_m - 1) / block_tile_m);

    int smem_size = MAX(STAGES * 128 * 32 * 2 * 2, 128 * 128 * 4);
    
    if (smem_size >= (48 << 10)) {
        CUDA_CHECK(
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                matmul),
                hipFuncAttributeMaxDynamicSharedMemorySize,
                smem_size
            )
        );
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup
    for (int i = 0; i < iterations / 20 + 1; ++i) {
        matmul<<<dimGrid, dimBlock, smem_size>>>(dA, dB, dC, M, N, K, alpha, beta);
    }
    hipDeviceSynchronize();

    hipEventRecord(start);
    for (int i = 0; i < iterations; ++i) {
        matmul<<<dimGrid, dimBlock, smem_size>>>(dA, dB, dC, M, N, K, alpha, beta);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    CUDA_CHECK(hipMemcpy(hC, dC, M * N * 2, hipMemcpyDeviceToHost));

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Running cost of CUDA kernel is " << double(ms) / iterations << "ms\n";
    std::cout << "TFLOPS: " << (float)M * N * K * 2 / (double(ms) / iterations) * 1e3 / 1e12 << "\n";

    if (checkResult(h_cublas, hC, M, N)) {
        puts("pass!");
    } else {
        puts("fail!");
    }

    free(hA);
    free(hB);
    free(hC);
    free(h_cublas);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}