#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <chrono>
#include <string>
#include <cassert>
#include <hipblas.h>
#include "includes/commons.cuh"

inline const char *cublas_get_error(hipblasStatus_t status) {
    switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
        return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "HIPBLAS_STATUS_NOT_INITIALIZED -- The cuBLAS library was not initialized.";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "HIPBLAS_STATUS_ALLOC_FAILED -- Resource allocation failed inside the cuBLAS library.";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "HIPBLAS_STATUS_INVALID_VALUE -- An unsupported value or parameter was passed to the function.";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "HIPBLAS_STATUS_ARCH_MISMATCH -- The function requires a feature absent from the device architecture.";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "HIPBLAS_STATUS_MAPPING_ERROR -- An access to GPU memory space failed.";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "HIPBLAS_STATUS_EXECUTION_FAILED -- The GPU program failed to execute.";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "HIPBLAS_STATUS_INTERNAL_ERROR -- An internal cuBLAS operation failed.";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
        return "HIPBLAS_STATUS_NOT_SUPPORTED -- The functionality requested is not supported.";
    case HIPBLAS_STATUS_UNKNOWN:
        return "HIPBLAS_STATUS_UNKNOWN -- An error was detected when checking the current licensing.";
    default:
        return "CUBLAS_ERROR -- <unknown>";
    }
}

inline bool cublas_is_error(hipblasStatus_t status) {
    return status != HIPBLAS_STATUS_SUCCESS;
}

inline hipblasStatus_t gemm(
    hipblasHandle_t handle,
    hipblasOperation_t transA, hipblasOperation_t transB,
    int m, int n, int k,
    const float* alpha,
    const half* A, int ldA,
    const half* B, int ldB,
    const float* beta,
    half* C, int ldC
) {
    return hipblasGemmEx(
        handle, transA, transB,
        m, n, k,
        reinterpret_cast<const float*>(alpha),
        reinterpret_cast<const __half*>(A), HIP_R_16F, ldA,
        reinterpret_cast<const __half*>(B), HIP_R_16F, ldB,
        reinterpret_cast<const float*>(beta),
        reinterpret_cast<__half*>(C), HIP_R_16F, ldC,
        HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );
}

void cublasMatmul(half *hC, half *hA, half *hB, int M, int N, int K) {
    half *dA, *dB, *dC;
    float alpha = 1.0f, beta = 0.0f;

    CUDA_CHECK(hipMalloc(&dA, M * K * 2));
    CUDA_CHECK(hipMalloc(&dB, K * N * 2));
    CUDA_CHECK(hipMalloc(&dC, M * N * 2));

    CUDA_CHECK(hipMemcpy(dA, hA, M * K * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, K * N * 2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dC, hC, M * N * 2, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    gemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, dA, K, dB, K, &beta, dC, M);
    CUDA_CHECK(hipMemcpy(hC, dC, M * N * 2, hipMemcpyDeviceToHost));

    hipblasDestroy(handle);
    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
}