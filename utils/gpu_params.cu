#include <iostream>
#include <hip/hip_runtime.h>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    checkCudaError(err, "Failed to get device count");

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        err = hipGetDeviceProperties(&deviceProp, dev);
        checkCudaError(err, "Failed to get device properties");

        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  Total Memory: " << (deviceProp.totalGlobalMem / (1024 * 1024)) << " MB" << std::endl;
        std::cout << "  Clock Rate: " << (deviceProp.clockRate / 1000) << " MHz" << std::endl;
        std::cout << "  L2 Cache Size: " << (deviceProp.l2CacheSize / 1024) << " KB" << std::endl;
        std::cout << "  Max Threads per SM: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Streaming Multiprocessors (SM) Count: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << std::endl;
    }

    return 0;
}