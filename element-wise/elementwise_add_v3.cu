#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

const int N = 1 << 25;
const int iterations = 2000;

__global__ void add_v3(float *a, float *b, float *result) {
    int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    float4 a_vec = *reinterpret_cast<float4 *>(&a[tid]);
    float4 b_vec = *reinterpret_cast<float4 *>(&b[tid]);
    float4 result_vec;
    result_vec.x = a_vec.x + b_vec.x;
    result_vec.y = a_vec.y + b_vec.y;
    result_vec.z = a_vec.z + b_vec.z;
    result_vec.w = a_vec.w + b_vec.w;
    *reinterpret_cast<float4 *>(&result[tid]) = result_vec;
}

bool check_result(float *a, float *b) {
    for (int i = 0; i < N; ++i) {
        if (a[i] != b[i]) {
            return false;
        }
    }
    return true;
}

int main() {
    float *host_a = (float *) malloc(N * sizeof(float));
    float *host_b = (float *) malloc(N * sizeof(float));
    float *host_result = (float *) malloc(N * sizeof(float));
    float *cpu_result = (float *) malloc(N * sizeof(float));

    float *device_a;
    float *device_b;
    float *device_result;

    hipMalloc((void **) &device_a, N * sizeof(float));
    hipMalloc((void **) &device_b, N * sizeof(float));
    hipMalloc((void **) &device_result, N * sizeof(float));

    for (int i = 0; i < N; ++i) {
        host_a[i] = i;
        host_b[i] = (N % (i + 1)) * 1.0f;
        cpu_result[i] = host_a[i] + host_b[i];
    }

    hipMemcpy(device_a, host_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, N * sizeof(float), hipMemcpyHostToDevice);
    int block_size = 256;
    int grid_size = (N - 1) / block_size + 1;

    dim3 Grid(grid_size / 4);
    dim3 Block(block_size);

    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iterations; ++i) {
        add_v3<<<Grid, Block>>>(device_a, device_b, device_result);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(host_result, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    if (!check_result(host_result, cpu_result)) {
        printf("Wrong Answer!\n");
    } else {
        printf("Success\n");
    }

    // Calculate memory bandwidth
    double total_data_transfer = 3 * N * sizeof(float); // 2 input arrays and 1 output array
    double bandwidth = (total_data_transfer / (1 << 30)) / (milliseconds / 1000.0 / iterations); // GB/s
    std::cout << "Elapsed time: " << milliseconds / 1000.0 << " seconds" << std::endl;
    std::cout << "Memory Bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
    free(host_a);
    free(host_b);
    free(host_result);
    free(cpu_result);

    return 0;
}