#include <iostream>
#include <hip/hip_runtime.h>

const int N = 1 << 5;
const int iterations = 1000;

__global__ void add_v1(float *a, float *b, float *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        result[tid] = a[tid] + b[tid];
    }
}

bool check_result(float *a, float *b) {
    for (int i = 0; i < N; ++i) {
        if (a[i] != b[i]) {
            return false;
        }
    }
    return true;
}

void check_cuda_error(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    float *host_a = (float *) malloc(N * sizeof(float));
    float *host_b = (float *) malloc(N * sizeof(float));
    float *host_result = (float *) malloc(N * sizeof(float));
    float *cpu_result = (float *) malloc(N * sizeof(float));

    float *device_a;
    float *device_b;
    float *device_result;

    hipError_t err;

    err = hipMalloc((void **) &device_a, N * sizeof(float));
    check_cuda_error(err, "hipMalloc device_a");
    err = hipMalloc((void **) &device_b, N * sizeof(float));
    check_cuda_error(err, "hipMalloc device_b");
    err = hipMalloc((void **) &device_result, N * sizeof(float));
    check_cuda_error(err, "hipMalloc device_result");

    for (int i = 0; i < N; ++i) {
        host_a[i] = i;
        host_b[i] = (N % (i + 1)) * 1.0f; // Cast to float
        cpu_result[i] = host_a[i] + host_b[i];
    }

    std::cout << "Initializing data\n";

    err = hipMemcpy(device_a, host_a, N * sizeof(float), hipMemcpyHostToDevice);
    check_cuda_error(err, "hipMemcpy host_a to device_a");
    err = hipMemcpy(device_b, host_b, N * sizeof(float), hipMemcpyHostToDevice);
    check_cuda_error(err, "hipMemcpy host_b to device_b");

    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;

    dim3 Grid(grid_size);
    dim3 Block(block_size);

    for (int i = 0; i < iterations; ++i) {
        add_v1<<<Grid, Block>>>(device_a, device_b, device_result);
        err = hipGetLastError();
        check_cuda_error(err, "Kernel launch");
    }

    err = hipMemcpy(host_result, device_result, N * sizeof(float), hipMemcpyDeviceToHost);
    check_cuda_error(err, "hipMemcpy device_result to host_result");

    if (!check_result(host_result, cpu_result)) {
        printf("Error\n");
        for (int i = 0; i < N; ++i) {
            std::cout << host_result[i] << " ";
        }
        std::cout << "\n";

        for (int i = 0; i < N; ++i) {
            std::cout << cpu_result[i] << " ";
        }
        std::cout << "\n";
    } else {
        printf("Success\n");
    }

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
    free(host_a);
    free(host_b);
    free(host_result);
    free(cpu_result);

    return 0;
}
